#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>
#include "random.h"
#include "helpers.h"

using namespace optix;

struct PerRayData_pathtrace
{
    float3 result;
    float3 radiance;
    float3 attenuation;
    float3 origin;
    float3 direction;
    unsigned int seed;
    int depth;
    int countEmitted;
    int done;
};

// Scene wide variables
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint2,         launch_index, rtLaunchIndex, );

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  samples_per_pixel, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );
rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  max_bounces, , );

rtBuffer<uchar4, 2>              output_buffer;


RT_PROGRAM void pathtrace_camera()
{
    size_t2 screen = output_buffer.size();

    float2 inv_screen = 1.0f/make_float2(screen) * 2.f;
    float2 pixel = (make_float2(launch_index)) * inv_screen - 1.f;

    float3 result = make_float3(0.0f);

    unsigned int seed = tea<16>(screen.x*launch_index.y+launch_index.x, frame_number);
	unsigned samples = samples_per_pixel;
    do 
    {

        float2 d = pixel;
        float3 ray_origin = eye;
        float3 ray_direction = normalize(d.x*U + d.y*V + W);

        // Initialze per-ray data
        PerRayData_pathtrace prd;
        prd.result = make_float3(0.f);
        prd.attenuation = make_float3(1.f);
        prd.countEmitted = true;
        prd.done = false;
        prd.seed = seed;
        prd.depth = 0;

        // Each iteration is a segment of the ray path.  The closest hit will
        // return new segments to be traced here.
        for(;;)
        {
            Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
            rtTrace(top_object, ray, prd);

            if(prd.done)
            {
                // We have hit the background or a luminaire
                prd.result += prd.radiance * prd.attenuation;
                break;
            }

            // Russian roulette termination 
            if(prd.depth >= rr_begin_depth)
            {
                float pcont = fmaxf(prd.attenuation);
                if(rnd(prd.seed) >= pcont)
                    break;
                prd.attenuation /= pcont;
            }

			if(prd.depth >= max_bounces)
			{
				break;
			}

            prd.depth++;
            prd.result += prd.radiance * prd.attenuation;

            // Update ray data for the next path segment
            ray_origin = prd.origin;
            ray_direction = prd.direction;
        }

        result += prd.result;
        seed = prd.seed;
    } while (--samples);


    float3 pixel_color = result/(samples_per_pixel);
	output_buffer[launch_index] = make_color(pixel_color);
	
}

RT_PROGRAM void exception()
{
    output_buffer[launch_index] = make_color(bad_color);
}