#include "hip/hip_runtime.h"
#include <optixu/optixu_math_namespace.h>
#include "commonStructs.h"
#include "random.h"

using namespace optix;

struct PerRayData_pathtrace
{
    float3 result;
    float3 radiance;
    float3 attenuation;
    float3 origin;
    float3 direction;
    unsigned int seed;
    int depth;
    int countEmitted;
    int done;
};

struct PerRayData_pathtrace_shadow
{
    bool inShadow;
};

// Scene wide variables
rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint,          launch_index, rtLaunchIndex, );

rtDeclareVariable(PerRayData_pathtrace, current_prd, rtPayload, );



//-----------------------------------------------------------------------------
//
//  Camera program -- main ray tracing loop
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  samples_per_pixel, , );
rtDeclareVariable(unsigned int,  rr_begin_depth, , );
rtDeclareVariable(unsigned int,  pathtrace_ray_type, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );
rtDeclareVariable(unsigned int,  max_bounces, , );

rtBuffer<float4>                radiosity_buffer;
rtBuffer<BasicLight>			lights;
rtBuffer<opVPL>					vpl_buffer;


RT_PROGRAM void pathtrace_entry()
{
    float3 result = make_float3(0.0f);
	opVPL vpl = vpl_buffer[launch_index];
    unsigned int seed = vpl.seed;
	unsigned int samplenum = samples_per_pixel;
	float3 ray_origin = vpl.pos;

    optix::Onb onb( vpl.normal );

	float raynum = 0;
    do 
    {
        // Initialze per-ray data
        PerRayData_pathtrace prd;
        prd.result = make_float3(0.f);
        prd.attenuation = vpl.color;
        prd.countEmitted = true;
        prd.done = false;
        prd.seed = seed;
        prd.depth = 0;

		float z1=rnd(seed);
		float z2=rnd(seed);
		float3 ray_direction;
		cosine_sample_hemisphere(z1, z2, ray_direction);
		onb.inverse_transform( ray_direction );

        // Each iteration is a segment of the ray path.  The closest hit will
        // return new segments to be traced here.
        for(;;)
        {
            Ray ray = make_Ray(ray_origin, ray_direction, pathtrace_ray_type, scene_epsilon, RT_DEFAULT_MAX);
            rtTrace(top_object, ray, prd);
			
            if(prd.done)
            {
                // We have hit the background or a luminaire
                prd.result += prd.radiance * prd.attenuation;
                break;
            }

			if(prd.depth >= max_bounces)
			{
				break;
			}

            // Russian roulette termination 
            if(prd.depth >= rr_begin_depth)
            {
                float pcont = fmaxf(prd.attenuation);
                if(rnd(prd.seed) >= pcont)
				{
					break;
				}
                prd.attenuation /= pcont;
            }
			raynum+=1;
            prd.depth++;
            prd.result += prd.radiance * prd.attenuation;

            // Update ray data for the next path segment
            ray_origin = prd.origin;
            ray_direction = prd.direction;
        }

        result += prd.result;
        seed = prd.seed;
    } while (--samplenum);
	

    float3 pixel_color = result/(samples_per_pixel);

	float3 light_color = make_float3(0.0f);
	unsigned int num_lights = lights.size();
	float shadowed = 1;

	for(int i = 0; i < num_lights; ++i)
    {
        const float3 light_pos = lights[i].pos;

        // Calculate properties of light sample (for area based pdf)
        const float  Ldist = length(light_pos - vpl.pos);
        const float3 L     = normalize(light_pos - vpl.pos);
        const float  nDl   = dot( vpl.normal, L );
		const float  LnDl  = dot( lights[i].normal, -L );

        // cast shadow ray
        if ( nDl > 0.0f  && LnDl > 0.0f )
        {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            Ray shadow_ray = make_Ray( vpl.pos, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon );
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow)
            {
				shadowed = 0;
                const float weight = nDl * LnDl * 30 / ( Ldist * Ldist);
                light_color += lights[i].color * weight * vpl.color;
            }
        }
    }
	radiosity_buffer[launch_index] = make_float4(light_color + pixel_color, shadowed);
}


//-----------------------------------------------------------------------------
//
//  Emissive surface closest-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3,        emission_color, , );

RT_PROGRAM void diffuseEmitter()
{
    current_prd.radiance = current_prd.countEmitted ? emission_color : make_float3(0.f);
    current_prd.done = true;
}


//-----------------------------------------------------------------------------
//
//  Lambertian surface closest-hit
//
//-----------------------------------------------------------------------------
rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3,     shading_normal,   attribute shading_normal, );
rtDeclareVariable(optix::Ray, pt_ray,           rtCurrentRay, );
rtDeclareVariable(float,      t_hit,            rtIntersectionDistance, );
rtDeclareVariable(float3,     Kd,				, );

RT_PROGRAM void diffuse()
{
    float3 world_shading_normal   = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal ) );
    float3 world_geometric_normal = normalize( rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal ) );
    float3 ffnormal = faceforward( world_shading_normal, -pt_ray.direction, world_geometric_normal );

    float3 hitpoint = pt_ray.origin + t_hit * pt_ray.direction;

    //
    // Generate a reflection ray.  This will be traced back in ray-gen.
    //
    current_prd.origin = hitpoint;

    float z1=rnd(current_prd.seed);
    float z2=rnd(current_prd.seed);
    float3 p;
    cosine_sample_hemisphere(z1, z2, p);
    optix::Onb onb( ffnormal );
    onb.inverse_transform( p );
    current_prd.direction = p;

    // NOTE: f/pdf = 1 since we are perfectly importance sampling lambertian
    // with cosine density.
    current_prd.attenuation = current_prd.attenuation * Kd;
    current_prd.countEmitted = false;

    //
    // Next event estimation (compute direct lighting).
    //
    unsigned int num_lights = lights.size();
    float3 result = make_float3(0.0f);

    for(int i = 0; i < num_lights; ++i)
    {
        const float3 light_pos = lights[i].pos;

        // Calculate properties of light sample (for area based pdf)
        const float  Ldist = length(light_pos - hitpoint);
        const float3 L     = normalize(light_pos - hitpoint);
        const float  nDl   = dot( ffnormal, L );
		const float  LnDl  = dot( lights[i].normal, -L );

        // cast shadow ray
        if ( nDl > 0.0f  && LnDl > 0.0f )
        {
            PerRayData_pathtrace_shadow shadow_prd;
            shadow_prd.inShadow = false;
            // Note: bias both ends of the shadow ray, in case the light is also present as geometry in the scene.
            Ray shadow_ray = make_Ray( hitpoint, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon );
            rtTrace(top_object, shadow_ray, shadow_prd);

            if(!shadow_prd.inShadow)
            {
                const float weight = nDl * LnDl * 30 / (Ldist * Ldist);
                result += lights[i].color * weight;
            }
        }
    }

    current_prd.radiance = result;
}


//-----------------------------------------------------------------------------
//
//  Shadow any-hit
//
//-----------------------------------------------------------------------------

rtDeclareVariable(PerRayData_pathtrace_shadow, current_prd_shadow, rtPayload, );

RT_PROGRAM void shadow()
{
    current_prd_shadow.inShadow = true;
    rtTerminateRay();
}


//-----------------------------------------------------------------------------
//
//  Exception program
//
//-----------------------------------------------------------------------------

RT_PROGRAM void exception()
{
    radiosity_buffer[launch_index] = make_float4(bad_color, 1.0f);
}


//-----------------------------------------------------------------------------
//
//  Miss program
//
//-----------------------------------------------------------------------------

rtDeclareVariable(float3, bg_color, , );

RT_PROGRAM void miss()
{
    current_prd.radiance = bg_color;
    current_prd.done = true;
}
