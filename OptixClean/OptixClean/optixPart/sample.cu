#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_world.h>
#include "helpers.h"
#include "commonStructs.h"
#include "random.h"

using namespace optix;

struct PerRayData
{
  float3 pos;
  float3 nor;
  float3 clr;
  float importance;
  int depth;
  unsigned int seed;
};

rtDeclareVariable(unsigned int,      sample_ray_type, , );
rtDeclareVariable(unsigned int,      sampleDirectlight_ray_type, , );
rtDeclareVariable(float,             scene_epsilon, , );
rtDeclareVariable(rtObject,          top_object, , );

rtDeclareVariable(PerRayData, prd, rtPayload, );

rtDeclareVariable(float3,       Ka, , );
rtDeclareVariable(float3,       Kd, , );
rtDeclareVariable(float3,       Ks, , );
rtDeclareVariable(float3,       Kr, , );
rtDeclareVariable(float,        phong_exp, , );


rtDeclareVariable(float3,        eye, , );
rtDeclareVariable(float3,        U, , );
rtDeclareVariable(float3,        V, , );
rtDeclareVariable(float3,        W, , );
rtDeclareVariable(float3,        bad_color, , );
rtDeclareVariable(float3,		 bg_color, , );
rtBuffer<float4>                 sample_pos_buffer;
rtBuffer<float4>                 sample_nor_buffer;
rtBuffer<float4>                 sample_clr_buffer;

rtDeclareVariable(unsigned int,  frame_number, , );

rtDeclareVariable(uint, launch_index, rtLaunchIndex, );
rtDeclareVariable(uint, launch_dim,   rtLaunchDim, );

// sample cam ray gen program
RT_PROGRAM void pinhole_camera_samCam()
{
  unsigned int seed = tea<16>(launch_index, frame_number);
  float2 d = make_float2(rnd(seed), rnd(seed));

  float3 ray_origin = eye;
  float3 ray_direction = normalize(d.x*U + d.y*V + W);
  
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, sample_ray_type, scene_epsilon, RT_DEFAULT_MAX);

  PerRayData newprd;
  newprd.importance = 1.f;
  newprd.depth = 0;
  newprd.seed = seed;

  rtTrace(top_object, ray, newprd);

  sample_pos_buffer[launch_index] = make_float4( newprd.pos.x, newprd.pos.y, newprd.pos.z, 1);
  sample_nor_buffer[launch_index] = make_float4( newprd.nor.x, newprd.nor.y, newprd.nor.z, 1);
  sample_clr_buffer[launch_index] = make_float4( newprd.clr.x, newprd.clr.y, newprd.clr.z, 1);
}


rtDeclareVariable(float3,     geometric_normal, attribute geometric_normal, );
rtDeclareVariable(float3,     shading_normal,   attribute shading_normal, );
rtDeclareVariable(optix::Ray, ray,              rtCurrentRay, );
rtDeclareVariable(float,      t_hit,            rtIntersectionDistance, );

// used to handle closest hit of camera sample and one bounce light sample
RT_PROGRAM void closest_hit_sample()
{
	float3 world_geo_normal = normalize(rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);

	float3 hit_point = ray.origin + t_hit * ray.direction;
	
	if(prd.depth<1){
		prd.depth += 1;
		float z1=rnd(prd.seed);
		float z2=rnd(prd.seed);
		float3 p;
		cosine_sample_hemisphere(z1, z2, p);
		optix::Onb onb( ffnormal );
		onb.inverse_transform( p );
		optix::Ray refl_ray = optix::make_Ray( hit_point, p, sample_ray_type, scene_epsilon, RT_DEFAULT_MAX );
		rtTrace(top_object, refl_ray, prd);
	}
	else{
		prd.pos = hit_point;
		prd.nor = ffnormal;
		prd.clr = Kd;
	}
}

__device__ int getRandSign(unsigned int& seed)
{
	return (rnd(seed)<0.5f)?1:-1;
}

rtBuffer<BasicLight>                 lights;

// sample light ray gen program
RT_PROGRAM void samdirect_light_raygen()
{
  unsigned int seed = tea<16>(launch_index, frame_number);

  BasicLight light = lights[0];
  float3 ray_origin = light.pos;
  
  float z1=rnd(seed);
  float z2=rnd(seed);
  float3 ray_direction;
  cosine_sample_hemisphere(z1, z2, ray_direction);
  optix::Onb onb( light.normal );
  onb.inverse_transform( ray_direction );
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, sampleDirectlight_ray_type, scene_epsilon, RT_DEFAULT_MAX);

  PerRayData newprd;
  newprd.importance = 1.f;
  newprd.depth = 0;
  newprd.seed = seed;

  rtTrace(top_object, ray, newprd);

  sample_pos_buffer[launch_index] = make_float4( newprd.pos.x, newprd.pos.y, newprd.pos.z, 1);
  sample_nor_buffer[launch_index] = make_float4( newprd.nor.x, newprd.nor.y, newprd.nor.z, 1);
  sample_clr_buffer[launch_index] = make_float4( newprd.clr.x, newprd.clr.y, newprd.clr.z, 1);
}

RT_PROGRAM void samlight_raygen()
{
  unsigned int seed = tea<16>(launch_index, frame_number);

  BasicLight light = lights[0];
  float3 ray_origin = light.pos;
  
  float z1=rnd(seed);
  float z2=rnd(seed);
  float3 ray_direction;
  cosine_sample_hemisphere(z1, z2, ray_direction);
  optix::Onb onb( light.normal );
  onb.inverse_transform( ray_direction );
  optix::Ray ray = optix::make_Ray(ray_origin, ray_direction, sample_ray_type, scene_epsilon, RT_DEFAULT_MAX);

  PerRayData newprd;
  newprd.importance = 1.f;
  newprd.depth = 0;
  newprd.seed = seed;

  rtTrace(top_object, ray, newprd);

  sample_pos_buffer[launch_index] = make_float4( newprd.pos.x, newprd.pos.y, newprd.pos.z, 1);
  sample_nor_buffer[launch_index] = make_float4( newprd.nor.x, newprd.nor.y, newprd.nor.z, 1);
  sample_clr_buffer[launch_index] = make_float4( newprd.clr.x, newprd.clr.y, newprd.clr.z, 1);
}

// sample point directly from light (Dirlight)
RT_PROGRAM void closest_hit_samplelight()
{
	float3 world_geo_normal = normalize(rtTransformNormal( RT_OBJECT_TO_WORLD, geometric_normal));
	float3 world_shade_normal = normalize(rtTransformNormal( RT_OBJECT_TO_WORLD, shading_normal));
	float3 ffnormal = faceforward(world_shade_normal, -ray.direction, world_geo_normal);
	float3 hit_point = ray.origin + t_hit * ray.direction;
	prd.depth += 1;
	prd.pos = hit_point;
	prd.nor = ffnormal;
	prd.clr = Kd;
}


// sample miss program
RT_PROGRAM void miss()
{
  prd.pos = bg_color;
  prd.nor = bg_color;
  prd.clr = bg_color;
}

// sample ray exception program
RT_PROGRAM void exception()
{
  const unsigned int code = rtGetExceptionCode();
  rtPrintf( "Caught exception 0x%X at launch index (%d)\n", code, launch_index );
  sample_pos_buffer[launch_index] = make_float4(bad_color.x, bad_color.y, bad_color.z, 1);
  sample_nor_buffer[launch_index] = make_float4(bad_color.x, bad_color.y, bad_color.z, 1);
  sample_clr_buffer[launch_index] = make_float4(bad_color.x, bad_color.y, bad_color.z, 1);
}