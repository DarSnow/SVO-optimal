#include "hip/hip_runtime.h"
#pragma once
#include <optixu/optixu_math_namespace.h>
#include "commonStructs.h"
#include "random.h"
using namespace optix;

struct PerRayData_pathtrace_shadow
{
    bool inShadow;
};


rtDeclareVariable(float,         scene_epsilon, , );
rtDeclareVariable(rtObject,      top_object, , );
rtDeclareVariable(uint,          launch_index, rtLaunchIndex, );

rtDeclareVariable(unsigned int,  frame_number, , );
rtDeclareVariable(unsigned int,  pathtrace_shadow_ray_type, , );
rtDeclareVariable(unsigned int,  sensor_size, , );

rtBuffer<opVPL>					vpl_buffer;
rtBuffer<float3>				sensor_pos_buffer;
rtBuffer<float3>				sensor_nor_buffer;
rtBuffer<float>					contri_buffer;

RT_PROGRAM void estimate_pdf_entry()
{
	opVPL vpl = vpl_buffer[launch_index];
	float3 sensor = sensor_pos_buffer[launch_index];
	float3 sensor_nor = sensor_nor_buffer[launch_index];

	float contribution = 0;
	for(int i=0;i<sensor_size;++i){
		PerRayData_pathtrace_shadow shadow_prd;
        shadow_prd.inShadow = false;
		const float3 L = normalize( sensor - vpl.pos);
		const float  Ldist = length( sensor - vpl.pos);
        Ray shadow_ray = make_Ray( vpl.pos, L, pathtrace_shadow_ray_type, scene_epsilon, Ldist - scene_epsilon );
        rtTrace(top_object, shadow_ray, shadow_prd);

        if(!shadow_prd.inShadow && dot(sensor_nor, -L) > 0)
        {
            contribution += dot( vpl.normal, L ) * dot(sensor_nor, -L) / ( Ldist * Ldist);
        }
	}
	contri_buffer[launch_index] = contribution / sensor_size;
}

RT_PROGRAM void exception(){
	
}